
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

	/*************************************************************************/
      
      __shared__ float ds_A [TILE_SIZE][TILE_SIZE];
      __shared__ float ds_B [TILE_SIZE][TILE_SIZE];

      int bx = blockIdx.x;
      int by = blockIdx.y;
      int tx = threadIdx.x;
      int ty = threadIdx.y;

      int Row = by * blockDim.y + ty;
      int Col = bx * blockDim.x + tx;
      float Pvalue = 0;

      for(int p = 0; p < (TILE_SIZE + k -1)/TILE_SIZE; p++){
 	if(p* TILE_SIZE + tx < k && Row < m){
	  ds_A[ty][tx] = A[Row*k + p*TILE_SIZE + tx];
	}
	else{
	  ds_A[ty][tx] = 0.0;
	}
	if (p*TILE_SIZE + ty < k && Col < n){
	  ds_B[ty][tx] = B[(p*TILE_SIZE + ty)*n + Col];
	}
	else{
	  ds_B[ty][tx] = 0.0;
	}
	__syncthreads();

	for(int i = 0; i < TILE_SIZE; i++){
	  Pvalue += ds_A[ty][i] * ds_B[i][tx];
	}
	
	__syncthreads();

	if(Row < m && Col < n){
	   C[Row*n + Col] = Pvalue;
	}
}
	   
      /*************************************************************************/
}

/********************************************************************
*
* You don't need to use 
*   transa, transb, alpha, beta, lda, ldb, and ldc
* in your code.
*
* They can be used in more complicated matrix multiplication 
* algorithms but not in this assignment.
* Feel free to explore these algorithms.
*
* You can get the complete score by only using
*   m, n, k, A, B, and C   
*
********************************************************************/


void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc){

    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    dim3 dimGrid((n+ BLOCK_SIZE -1)/BLOCK_SIZE+1, (m+ BLOCK_SIZE + 1)/BLOCK_SIZE+1,1);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);

    mysgemm<<<dimGrid, dimBlock>>>(m,n,k,A,B,C);

}

